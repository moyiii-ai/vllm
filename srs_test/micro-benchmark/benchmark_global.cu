// benchmark_global_ldst_v4.cu
// Bidirectional P2P benchmark using SM-driven ld/st.volatile.global.v4.u32
// - Keeps previous "various sizes" bidirectional read/write tests
// - Adds whole-pass warm-up (run entire suite once, discard), then measured pass
// - Adds "All-Reduce-like" sizes (prefill/decode) and measures bidirectional read/write
//
// Build example (A100):
//   nvcc -O2 -arch=compute_80 -code=sm_80 -o benchmark_global_ldst_v4 benchmark_global_ldst_v4.cu
//
// Notes:
// - This benchmark uses inline PTX vectorized 16B loads/stores to better mimic NCCL's style.
// - Uses UVA + peer access; each GPU has two local buffers: send (source) and recv (destination).
// - Each kernel loops with "repeat" to increase runtime and stabilize timing.
// - We accumulate a checksum and use memory clobbers to avoid DCE and force real traffic.

#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <thread>
#include <cstdlib>
#include <cstdint>
#include <atomic>
#include <cstring>

#define CHECK(cmd) do { \
    hipError_t e__ = (cmd); \
    if (e__ != hipSuccess) { \
        std::cerr << "CUDA Error: " << hipGetErrorString(e__) \
                  << " (code " << int(e__) << ") at " << __FILE__ << ":" << __LINE__ << std::endl; \
        std::exit(EXIT_FAILURE); \
    } \
} while(0)

struct DeviceContext {
    int devId;
    // Local buffers:
    // - buf_send: data this GPU will read from (local) or send to peer
    // - buf_recv: data this GPU will write into (local) or receive from peer
    void* buf_send = nullptr;  // 16B aligned
    void* buf_recv = nullptr;  // 16B aligned
};

// Simple filler to touch memory (float)
__global__ void fillPatternF32(float* p, size_t n, float base) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    size_t stride = gridDim.x * blockDim.x;
    for (size_t idx = i; idx < n; idx += stride) {
        p[idx] = base + float(idx % 1024) * 0.001f;
    }
}

// Kernel: peer READ (this GPU loads from peer and stores into its local buffer)
// Vectorized 16B transactions using ld.volatile.global.v4.u32 and st.volatile.global.v4.u32
__global__ void peerReadKernelV4(uint32_t* __restrict__ dst_local_u32,
                                 const uint32_t* __restrict__ src_peer_u32,
                                 size_t n_vec4, int repeat, unsigned long long* checksum)
{
    // n_vec4 is the number of 16-byte vectors (v4.u32)
    const size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t stride = gridDim.x * blockDim.x;

    // Per-thread checksum to avoid DCE
    unsigned long long acc = 0ull;

    // Pointers as vector-of-4 u32 (16B per element)
    for (int r = 0; r < repeat; ++r) {
        for (size_t i = tid; i < n_vec4; i += stride) {
            // Compute 16B base addresses (as u32* + 4*i)
            const uint32_t* src_ptr = src_peer_u32 + (i << 2);
            uint32_t* dst_ptr       = dst_local_u32 + (i << 2);

            // Load 16B from peer (volatile)
            uint32_t x0, x1, x2, x3;
            asm volatile(
                "ld.volatile.global.v4.u32 {%0,%1,%2,%3}, [%4];\n"
                : "=r"(x0), "=r"(x1), "=r"(x2), "=r"(x3)
                : "l"(src_ptr)
                : "memory");

            // Store 16B into local (volatile write-through to be conservative)
            asm volatile(
                "st.volatile.global.v4.u32 [%0], {%1,%2,%3,%4};\n"
                :
                : "l"(dst_ptr), "r"(x0), "r"(x1), "r"(x2), "r"(x3)
                : "memory");

            // Mix into checksum
            acc += (unsigned long long)x0 + x1 + x2 + x3;
        }
        __syncthreads();
    }

    // Ensure global visibility (system scope)
    __threadfence_system();

    // Reduce per-thread contribution
    if (tid == 0) {
        atomicAdd(checksum, acc);
    }
}

// Kernel: peer WRITE (this GPU loads from its local buffer and stores into peer buffer)
__global__ void peerWriteKernelV4(uint32_t* __restrict__ dst_peer_u32,
                                  const uint32_t* __restrict__ src_local_u32,
                                  size_t n_vec4, int repeat, unsigned long long* checksum)
{
    const size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t stride = gridDim.x * blockDim.x;

    unsigned long long acc = 0ull;

    for (int r = 0; r < repeat; ++r) {
        for (size_t i = tid; i < n_vec4; i += stride) {
            const uint32_t* src_ptr = src_local_u32 + (i << 2);
            uint32_t* dst_ptr       = dst_peer_u32  + (i << 2);

            // Load 16B locally (volatile to avoid caching artifacts)
            uint32_t x0, x1, x2, x3;
            asm volatile(
                "ld.volatile.global.v4.u32 {%0,%1,%2,%3}, [%4];\n"
                : "=r"(x0), "=r"(x1), "=r"(x2), "=r"(x3)
                : "l"(src_ptr)
                : "memory");

            // Store 16B to peer (volatile)
            asm volatile(
                "st.volatile.global.v4.u32 [%0], {%1,%2,%3,%4};\n"
                :
                : "l"(dst_ptr), "r"(x0), "r"(x1), "r"(x2), "r"(x3)
                : "memory");

            acc += (unsigned long long)x0 + x1 + x2 + x3;
        }
        __syncthreads();
    }

    __threadfence_system();

    if (tid == 0) {
        atomicAdd(checksum, acc);
    }
}

static inline void initDevice(DeviceContext& ctx, size_t maxBytes) {
    CHECK(hipSetDevice(ctx.devId));
    // Allocate 16B-aligned buffers (cudaMalloc is sufficiently aligned)
    CHECK(hipMalloc(&ctx.buf_send, maxBytes));
    CHECK(hipMalloc(&ctx.buf_recv, maxBytes));

    // Touch memory with a pattern (float-wise) to avoid zero-pages / lazy mapping effects
    const size_t nFloat = maxBytes / sizeof(float);
    const int threads = 256, blocks = 256;
    fillPatternF32<<<blocks, threads>>>((float*)ctx.buf_send, nFloat, 1.0f + ctx.devId * 10.0f);
    fillPatternF32<<<blocks, threads>>>((float*)ctx.buf_recv, nFloat, 0.0f);
    CHECK(hipGetLastError());
    CHECK(hipDeviceSynchronize());
}

static inline void cleanupDevice(DeviceContext& ctx) {
    CHECK(hipSetDevice(ctx.devId));
    if (ctx.buf_send) CHECK(hipFree(ctx.buf_send));
    if (ctx.buf_recv) CHECK(hipFree(ctx.buf_recv));
}

// Return GB/s for a single kernel launch (read variant)
double runOneRead(int currentDev,
                  void* localDst, const void* peerSrc,
                  size_t bytes, int repeat,
                  int blocks, int threads)
{
    CHECK(hipSetDevice(currentDev));

    // Count number of 16B vectors
    size_t n_vec4 = bytes / 16;
    // Device checksum buffer
    unsigned long long* d_chk = nullptr;
    CHECK(hipMalloc(&d_chk, sizeof(unsigned long long)));
    CHECK(hipMemset(d_chk, 0, sizeof(unsigned long long)));

    hipEvent_t start, stop;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));

    CHECK(hipEventRecord(start));
    peerReadKernelV4<<<blocks, threads>>>(
        (uint32_t*)localDst, (const uint32_t*)peerSrc, n_vec4, repeat, d_chk);
    CHECK(hipGetLastError());
    CHECK(hipEventRecord(stop));
    CHECK(hipEventSynchronize(stop));

    float ms = 0.0f;
    CHECK(hipEventElapsedTime(&ms, start, stop));

    unsigned long long hostChk = 0;
    CHECK(hipMemcpy(&hostChk, d_chk, sizeof(unsigned long long), hipMemcpyDeviceToHost));
    (void)hostChk;

    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(stop));
    CHECK(hipFree(d_chk));

    // Cross-GPU traffic (bytes over the link) = bytes * repeat (one remote load per 16B)
    const double gb = (double(bytes) * repeat) / 1e9;
    const double sec = ms / 1e3;
    return gb / (sec > 0 ? sec : 1e-12);
}

// Return GB/s for a single kernel launch (write variant)
double runOneWrite(int currentDev,
                   void* peerDst, const void* localSrc,
                   size_t bytes, int repeat,
                   int blocks, int threads)
{
    CHECK(hipSetDevice(currentDev));

    size_t n_vec4 = bytes / 16;
    unsigned long long* d_chk = nullptr;
    CHECK(hipMalloc(&d_chk, sizeof(unsigned long long)));
    CHECK(hipMemset(d_chk, 0, sizeof(unsigned long long)));

    hipEvent_t start, stop;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));

    CHECK(hipEventRecord(start));
    peerWriteKernelV4<<<blocks, threads>>>(
        (uint32_t*)peerDst, (const uint32_t*)localSrc, n_vec4, repeat, d_chk);
    CHECK(hipGetLastError());
    CHECK(hipEventRecord(stop));
    CHECK(hipEventSynchronize(stop));

    float ms = 0.0f;
    CHECK(hipEventElapsedTime(&ms, start, stop));

    unsigned long long hostChk = 0;
    CHECK(hipMemcpy(&hostChk, d_chk, sizeof(unsigned long long), hipMemcpyDeviceToHost));
    (void)hostChk;

    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(stop));
    CHECK(hipFree(d_chk));

    // Cross-GPU traffic (bytes over the link) = bytes * repeat (one remote store per 16B)
    const double gb = (double(bytes) * repeat) / 1e9;
    const double sec = ms / 1e3;
    return gb / (sec > 0 ? sec : 1e-12);
}

// Heuristic to choose repeat count so each kernel runs ~10–30ms for stability.
int chooseRepeat(size_t bytes) {
    // Tune these as needed. The smaller the message, the larger the repeat.
    if (bytes <= (1u<<20))   return 2000;   // 1 MB
    if (bytes <= (8u<<20))   return 800;
    if (bytes <= (64u<<20))  return 200;
    if (bytes <= (256u<<20)) return 80;
    return 40;
}

// Run a whole pass (all sizes) for bidirectional READ; if print=false, treat as warm-up.
void runPassBiRead(const DeviceContext& d0, const DeviceContext& d1,
                   const std::vector<size_t>& sizes, bool print)
{
    const int blocks = 256, threads = 256;
    for (auto bytes : sizes) {
        // Require 16B multiple
        size_t bytes16 = (bytes / 16) * 16;
        if (bytes16 == 0) continue;
        const int repeat = chooseRepeat(bytes16);
        double bw0 = 0.0, bw1 = 0.0;
        std::thread t0([&]{ bw0 = runOneRead(d0.devId, d0.buf_recv, d1.buf_send, bytes16, repeat, blocks, threads); });
        std::thread t1([&]{ bw1 = runOneRead(d1.devId, d1.buf_recv, d0.buf_send, bytes16, repeat, blocks, threads); });
        t0.join(); t1.join();
        if (print) {
            std::cout << "Read: dev0<-dev1 | Size = " << (bytes16 / 1024.0 / 1024.0)
                      << " MB | Bandwidth = " << bw0 << " GB/s\n";
            std::cout << "Read: dev1<-dev0 | Size = " << (bytes16 / 1024.0 / 1024.0)
                      << " MB | Bandwidth = " << bw1 << " GB/s\n";
        }
    }
}

// Run a whole pass (all sizes) for bidirectional WRITE; if print=false, treat as warm-up.
void runPassBiWrite(const DeviceContext& d0, const DeviceContext& d1,
                    const std::vector<size_t>& sizes, bool print)
{
    const int blocks = 256, threads = 256;
    for (auto bytes : sizes) {
        size_t bytes16 = (bytes / 16) * 16;
        if (bytes16 == 0) continue;
        const int repeat = chooseRepeat(bytes16);
        double bw01 = 0.0, bw10 = 0.0;
        std::thread t0([&]{ bw01 = runOneWrite(d0.devId, d1.buf_recv, d0.buf_send, bytes16, repeat, blocks, threads); });
        std::thread t1([&]{ bw10 = runOneWrite(d1.devId, d0.buf_recv, d1.buf_send, bytes16, repeat, blocks, threads); });
        t0.join(); t1.join();
        if (print) {
            std::cout << "Write: dev0->dev1 | Size = " << (bytes16 / 1024.0 / 1024.0)
                      << " MB | Bandwidth = " << bw01 << " GB/s\n";
            std::cout << "Write: dev1->dev0 | Size = " << (bytes16 / 1024.0 / 1024.0)
                      << " MB | Bandwidth = " << bw10 << " GB/s\n";
        }
    }
}

// Convenience to run: warm-up pass then measured pass
void runWithWarmup(const char* title,
                   void (*passFn)(const DeviceContext&, const DeviceContext&, const std::vector<size_t>&, bool),
                   const DeviceContext& d0, const DeviceContext& d1,
                   const std::vector<size_t>& sizes)
{
    passFn(d0, d1, sizes, /*print=*/false);
    std::cout << title << "\n";
    passFn(d0, d1, sizes, /*print=*/true);
}

int main() {
    // Require at least two devices
    int deviceCount = 0;
    CHECK(hipGetDeviceCount(&deviceCount));
    if (deviceCount < 2) {
        std::cerr << "Need at least 2 GPUs.\n";
        return 0;
    }
    const int dev0 = 0, dev1 = 1;

    // Check and enable P2P
    int can01 = 0, can10 = 0;
    CHECK(hipDeviceCanAccessPeer(&can01, dev0, dev1));
    CHECK(hipDeviceCanAccessPeer(&can10, dev1, dev0));
    if (!can01 || !can10) {
        std::cerr << "Peer access not supported between " << dev0 << " and " << dev1 << "\n";
        return 0;
    }
    CHECK(hipSetDevice(dev0));
    (void)hipDeviceEnablePeerAccess(dev1, 0); // ignore if already enabled
    CHECK(hipSetDevice(dev1));
    (void)hipDeviceEnablePeerAccess(dev0, 0);

    // Baseline sizes (bytes): 1MB, 8MB, 64MB, 256MB
    std::vector<size_t> sizes_baseline = {
        size_t(1)  << 20,
        size_t(8)  << 20,
        size_t(64) << 20,
        size_t(256)<< 20
    };

    // All-Reduce-like sizes (bytes):
    // - decode (per token per rank) tends to be small (tens of KB); choose 64KB, 256KB
    // - prefill (sequence/batch aggregated) can be multi-MB; choose 8MB, 16MB
    // You can adjust these to your exact TP topology and dtype.
    std::vector<size_t> sizes_allreduce_decode = {
        size_t(8) << 10,   // 8 KB
        size_t(64)  << 10,  // 64 KB
    };
    std::vector<size_t> sizes_allreduce_prefill = {
        size_t(8)  << 20,   // 8 MB
        size_t(16) << 20    // 16 MB
    };

    // Allocate maximum buffer we will use (align to largest)
    size_t maxBytes = 0;
    auto updMax = [&](const std::vector<size_t>& v){ for (auto b : v) if (b > maxBytes) maxBytes = b; };
    updMax(sizes_baseline);
    updMax(sizes_allreduce_decode);
    updMax(sizes_allreduce_prefill);

    DeviceContext d0{dev0}, d1{dev1};
    initDevice(d0, maxBytes);
    initDevice(d1, maxBytes);

    // 1) Baseline: Bidirectional WRITE, then READ (each has a whole-pass warm-up)
    runWithWarmup("=== Bidirectional Write (various sizes) ===", runPassBiWrite, d0, d1, sizes_baseline);
    runWithWarmup("=== Bidirectional Read  (various sizes) ===", runPassBiRead , d0, d1, sizes_baseline);
    printf("\n\n");
    // 2) All-Reduce-like: decode sizes
    runWithWarmup("=== Bidirectional Write (All-Reduce decode sizes) ===", runPassBiWrite, d0, d1, sizes_allreduce_decode);
    runWithWarmup("=== Bidirectional Read  (All-Reduce decode sizes) ===", runPassBiRead , d0, d1, sizes_allreduce_decode);
    printf("\n\n");
    // 3) All-Reduce-like: prefill sizes
    runWithWarmup("=== Bidirectional Write (All-Reduce prefill sizes) ===", runPassBiWrite, d0, d1, sizes_allreduce_prefill);
    runWithWarmup("=== Bidirectional Read  (All-Reduce prefill sizes) ===", runPassBiRead , d0, d1, sizes_allreduce_prefill);

    cleanupDevice(d0);
    cleanupDevice(d1);
    return 0;
}
