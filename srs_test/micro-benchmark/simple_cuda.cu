#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>

#define CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            fprintf(stderr, "Error: %s in %s:%d\n", hipGetErrorString(err), __FILE__, __LINE__); \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

int main(int argc, char* argv[]) {
    if (argc != 2) { fprintf(stderr, "Usage: %s [read|write]\n", argv[0]); exit(EXIT_FAILURE); }
    bool isReadMode = (strcmp(argv[1], "read") == 0);
    if (!isReadMode && strcmp(argv[1], "write") != 0) { fprintf(stderr, "Invalid mode: use 'read'/'write'\n"); exit(EXIT_FAILURE); }

    const size_t dataSize = 8ULL * 1024 * 1024 * 1024; // 8GB
    const double dataSizeGB = static_cast<double>(dataSize) / (1024 * 1024 * 1024);

    // GPU count & P2P check
    int deviceCount; CHECK(hipGetDeviceCount(&deviceCount));
    if (deviceCount < 2) { fprintf(stderr, "Need at least 2 GPUs\n"); exit(EXIT_FAILURE); }
    int canAccess0to1, canAccess1to0;
    CHECK(hipSetDevice(0)); CHECK(hipDeviceCanAccessPeer(&canAccess0to1, 0, 1));
    CHECK(hipSetDevice(1)); CHECK(hipDeviceCanAccessPeer(&canAccess1to0, 1, 0));
    if (!canAccess0to1 || !canAccess1to0) { fprintf(stderr, "P2P not supported\n"); exit(EXIT_FAILURE); }

    // Enable P2P & allocate memory
    CHECK(hipSetDevice(0)); CHECK(hipDeviceEnablePeerAccess(1, 0));
    CHECK(hipSetDevice(1)); CHECK(hipDeviceEnablePeerAccess(0, 0)); CHECK(hipSetDevice(0));
    double *d0, *d1; CHECK(hipMalloc(&d0, dataSize));
    CHECK(hipSetDevice(1)); CHECK(hipMalloc(&d1, dataSize));

    // Initialize memory with pattern
    unsigned char pattern = isReadMode ? 0xBB : 0xAA;
    CHECK(hipMemset(d1, pattern, dataSize));
    if (!isReadMode) { CHECK(hipSetDevice(0)); CHECK(hipMemset(d0, pattern, dataSize)); }
    CHECK(hipSetDevice(0));

    // Create stream & events
    hipStream_t stream; hipEvent_t start, stop;
    CHECK(hipStreamCreate(&stream));
    CHECK(hipEventCreate(&start)); CHECK(hipEventCreate(&stop));

    printf("Initialization done! Start counter polling and press...\n");
    getchar();

    // Timed P2P transfer
    CHECK(hipEventRecord(start, stream));
    if (isReadMode) {
        CHECK(hipMemcpyPeerAsync(d0, 0, d1, 1, dataSize, stream));
    } else {
        CHECK(hipMemcpyPeerAsync(d1, 1, d0, 0, dataSize, stream));
    }
    CHECK(hipStreamSynchronize(stream));
    CHECK(hipEventRecord(stop, stream));
    CHECK(hipEventSynchronize(stop));

    printf("All tests done! Stop counter polling and press...\n");
    getchar();

    // Calculate time & throughput
    float elapsedMs;
    CHECK(hipEventElapsedTime(&elapsedMs, start, stop));
    double elapsedSec = elapsedMs / 1000.0;
    double throughput = dataSizeGB / elapsedSec;

    // Print results - removed reference to non-existent 'verified' variable
    printf("%s transfer completed\n", isReadMode ? "Read" : "Write");
    printf("Size: %.2f GB | Time: %.2f ms | Throughput: %.4f GB/s\n", 
           dataSizeGB, elapsedMs, throughput);

    // Cleanup
    CHECK(hipFree(d0)); 
    CHECK(hipSetDevice(1)); 
    CHECK(hipFree(d1));
    CHECK(hipDeviceDisablePeerAccess(0)); 
    CHECK(hipSetDevice(0)); 
    CHECK(hipDeviceDisablePeerAccess(1));
    hipStreamDestroy(stream); 
    hipEventDestroy(start); 
    hipEventDestroy(stop);

    return 0;
}
