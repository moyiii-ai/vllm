// micro_benchmark.cu
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <thread>
#include <cstdio>

#define CHECK(call)                                                       \
    do {                                                                  \
        hipError_t err__ = (call);                                       \
        if (err__ != hipSuccess) {                                       \
            std::cerr << "CUDA Error: " << hipGetErrorString(err__)      \
                      << " (code " << err__ << ") at " << __FILE__ << ":" \
                      << __LINE__ << std::endl;                           \
            std::exit(EXIT_FAILURE);                                      \
        }                                                                 \
    } while (0)

// Per-GPU context: two buffers (send/recv) to avoid buffer conflicts in bidirectional tests.
struct DeviceContext {
    int devId;
    char* buf_send = nullptr;
    char* buf_recv = nullptr;
};

// Allocate per-GPU buffers with given capacity (bytes).
void initDevice(DeviceContext& ctx, size_t capacityBytes) {
    CHECK(hipSetDevice(ctx.devId));
    CHECK(hipMalloc(&ctx.buf_send, capacityBytes));
    CHECK(hipMalloc(&ctx.buf_recv, capacityBytes));
}

// Free per-GPU buffers.
void cleanupDevice(DeviceContext& ctx) {
    CHECK(hipSetDevice(ctx.devId));
    if (ctx.buf_send) CHECK(hipFree(ctx.buf_send));
    if (ctx.buf_recv) CHECK(hipFree(ctx.buf_recv));
}

// Single-direction P2P copy loop (srcDev -> dstDev), timed with CUDA events.
// Returns average bandwidth in GB/s over 'iters' copies of 'bytes'.
double runSingleDirection(const void* src, int srcDev, void* dst, int dstDev,
                          size_t bytes, int iters) {
    CHECK(hipSetDevice(srcDev));
    hipStream_t stream;
    CHECK(hipStreamCreate(&stream));

    hipEvent_t start, stop;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));

    // Timed loop
    CHECK(hipEventRecord(start, stream));
    for (int i = 0; i < iters; i++) {
        CHECK(hipMemcpyPeerAsync(dst, dstDev, src, srcDev, bytes, stream));
    }
    CHECK(hipEventRecord(stop, stream));
    CHECK(hipEventSynchronize(stop));

    float ms = 0.0f;
    CHECK(hipEventElapsedTime(&ms, start, stop));

    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(stop));
    CHECK(hipStreamDestroy(stream));

    // Avg time per copy (ms)
    const double avg_ms = ms / static_cast<double>(iters);
    // GB/s = (bytes per copy) / (seconds per copy)
    const double gb = static_cast<double>(bytes) / 1e9;
    const double secs = avg_ms / 1000.0;
    return gb / secs;
}

// Helper: run a lambda twice: first as warm-up (no print), then real (print).
template <typename F>
void runWithSingleWarmup(F&& f) {
    f(false);  // warm-up only
    f(true);   // measured run with printing
}

// Bidirectional "write" test (semantics: each GPU writes to the peer).
// Two threads run in parallel: dev0: send->dev1.recv, dev1: send->dev0.recv
void runSimultaneousWrite(const DeviceContext& ctx0, const DeviceContext& ctx1,
                          size_t bytes, int iters, bool print) {
    double bw01 = 0.0, bw10 = 0.0;

    std::thread t0([&]{
        bw01 = runSingleDirection(ctx0.buf_send, ctx0.devId,
                                  ctx1.buf_recv, ctx1.devId,
                                  bytes, iters);
    });
    std::thread t1([&]{
        bw10 = runSingleDirection(ctx1.buf_send, ctx1.devId,
                                  ctx0.buf_recv, ctx0.devId,
                                  bytes, iters);
    });
    t0.join();
    t1.join();

    if (print) {
        std::cout << "Write: dev0->dev1 | Size = " << (bytes / 1024.0 / 1024.0)
                  << " MB | Bandwidth = " << bw01 << " GB/s\n";
        std::cout << "Write: dev1->dev0 | Size = " << (bytes / 1024.0 / 1024.0)
                  << " MB | Bandwidth = " << bw10 << " GB/s\n";
    }
}

// Bidirectional "read" test (semantics: each GPU pulls data from the peer).
// Two threads run in parallel: dev0 reads from dev1 (src=dev1.send -> dst=dev0.recv), and vice versa.
void runSimultaneousRead(const DeviceContext& ctx0, const DeviceContext& ctx1,
                         size_t bytes, int iters, bool print) {
    double bw0reads = 0.0, bw1reads = 0.0;

    std::thread t0([&]{
        bw0reads = runSingleDirection(ctx1.buf_send, ctx1.devId,
                                      ctx0.buf_recv, ctx0.devId,
                                      bytes, iters);
    });
    std::thread t1([&]{
        bw1reads = runSingleDirection(ctx0.buf_send, ctx0.devId,
                                      ctx1.buf_recv, ctx1.devId,
                                      bytes, iters);
    });
    t0.join();
    t1.join();

    if (print) {
        std::cout << "Read: dev0<-dev1 | Size = " << (bytes / 1024.0 / 1024.0)
                  << " MB | Bandwidth = " << bw0reads << " GB/s\n";
        std::cout << "Read: dev1<-dev0 | Size = " << (bytes / 1024.0 / 1024.0)
                  << " MB | Bandwidth = " << bw1reads << " GB/s\n";
    }
}

// ===== Approximate all-reduce scale for LLaMA-3.1-8B =====
// We model only the message size/iteration pattern, not a real collective.
// Settings below are approximate, intended for relative bandwidth comparison.
struct AllReduceConfig {
    int   hidden_dim     = 4096;  // LLaMA-8B approx
    int   num_layers     = 32;
    int   ops_per_layer  = 2;     // typical: 2 all-reduce per layer (attn out + MLP out)
    int   dtype_bytes    = 2;     // FP16/BF16
    int   decode_tokens  = 128;   // number of generated tokens to simulate
    int   prefill_seqlen = 2048;  // typical prefill length
};

// Simulate decode: per-copy size = hidden_dim * dtype_bytes (≈8KB).
// Iterations = decode_tokens * num_layers * ops_per_layer.
void simulateDecode(const DeviceContext& ctx0, const DeviceContext& ctx1,
                    const AllReduceConfig& cfg) {
    const size_t bytes_per_copy = static_cast<size_t>(cfg.hidden_dim) * cfg.dtype_bytes; // ~8KB
    const int iters = cfg.decode_tokens * cfg.num_layers * cfg.ops_per_layer;

    runWithSingleWarmup([&](bool print){
        if (print) {
            std::cout << "\n=== Simulated All-Reduce Scale: Decode ===\n";
            std::cout << "per-copy size ≈ " << (bytes_per_copy / 1024.0) << " KB"
                      << ", iterations = " << iters << "\n";
            std::cout << "-- Bidirectional Write --\n";
        }
        runSimultaneousWrite(ctx0, ctx1, bytes_per_copy, iters, print);
        if (print) std::cout << "-- Bidirectional Read --\n";
        runSimultaneousRead(ctx0, ctx1, bytes_per_copy, iters, print);
    });
}

// Simulate prefill: per-copy size = seqlen * hidden_dim * dtype_bytes (≈16MB for 2048×4096×2).
// Iterations = num_layers * ops_per_layer (per forward pass).
void simulatePrefill(const DeviceContext& ctx0, const DeviceContext& ctx1,
                     const AllReduceConfig& cfg) {
    const size_t bytes_per_copy =
        static_cast<size_t>(cfg.prefill_seqlen) * cfg.hidden_dim * cfg.dtype_bytes; // ~16MB
    const int iters = cfg.num_layers * cfg.ops_per_layer; // per forward

    runWithSingleWarmup([&](bool print){
        if (print) {
            std::cout << "\n=== Simulated All-Reduce Scale: Prefill ===\n";
            std::cout << "per-copy size ≈ " << (bytes_per_copy / 1024.0 / 1024.0) << " MB"
                      << ", iterations = " << iters << "\n";
            std::cout << "-- Bidirectional Write --\n";
        }
        runSimultaneousWrite(ctx0, ctx1, bytes_per_copy, iters, print);
        if (print) std::cout << "-- Bidirectional Read --\n";
        runSimultaneousRead(ctx0, ctx1, bytes_per_copy, iters, print);
    });
}

int main() {
    // Pick two devices (0 and 1 by default).
    int numDevices = 0;
    CHECK(hipGetDeviceCount(&numDevices));
    if (numDevices < 2) {
        std::cerr << "Need at least 2 GPUs for this benchmark.\n";
        return 0;
    }
    const int dev0 = 0, dev1 = 1;

    int canAccess01 = 0, canAccess10 = 0;
    CHECK(hipDeviceCanAccessPeer(&canAccess01, dev0, dev1));
    CHECK(hipDeviceCanAccessPeer(&canAccess10, dev1, dev0));
    if (!canAccess01 || !canAccess10) {
        std::cerr << "Peer access not supported between GPUs " << dev0
                  << " and " << dev1 << "\n";
        return 0;
    }

    // Enable peer access on both devices.
    CHECK(hipSetDevice(dev0)); CHECK(hipDeviceEnablePeerAccess(dev1, 0));
    CHECK(hipSetDevice(dev1)); CHECK(hipDeviceEnablePeerAccess(dev0, 0));

    // Allocate per-GPU buffers sized for the largest test (max of base sizes and prefill size).
    std::vector<size_t> base_sizes = { size_t(1) << 20, size_t(8) << 20, size_t(64) << 20, size_t(256) << 20 };

    AllReduceConfig cfg; // defaults as defined above
    const size_t prefill_bytes = static_cast<size_t>(cfg.prefill_seqlen) * cfg.hidden_dim * cfg.dtype_bytes;

    size_t max_needed = 0;
    for (auto b : base_sizes) max_needed = std::max(max_needed, b);
    max_needed = std::max(max_needed, prefill_bytes);

    DeviceContext ctx0{dev0}, ctx1{dev1};
    initDevice(ctx0, max_needed);
    initDevice(ctx1, max_needed);

    // ===== Base tests: bidirectional write/read for multiple sizes =====
    runWithSingleWarmup([&](bool print){
        if (print) std::cout << "=== Simultaneous Write Test ===\n";
        for (auto bytes : base_sizes) {
            // More iters for small sizes to improve timing stability
            const int iters = (bytes <= (8u << 20)) ? 500 : 100;
            runSimultaneousWrite(ctx0, ctx1, bytes, iters, print);
        }
    });

    runWithSingleWarmup([&](bool print){
        if (print) std::cout << "\n=== Simultaneous Read Test ===\n";
        for (auto bytes : base_sizes) {
            const int iters = (bytes <= (8u << 20)) ? 500 : 100;
            runSimultaneousRead(ctx0, ctx1, bytes, iters, print);
        }
    });

    // ===== Simulate LLaMA-8B all-reduce scales =====
    simulateDecode (ctx0, ctx1, cfg);   // decode: ~8KB per copy, many iterations
    simulatePrefill(ctx0, ctx1, cfg);   // prefill: ~16MB per copy, fewer iterations

    cleanupDevice(ctx0);
    cleanupDevice(ctx1);
    return 0;
}
