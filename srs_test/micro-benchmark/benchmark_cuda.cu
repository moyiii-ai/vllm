#include <hip/hip_runtime.h>
#include <iostream>
#include <thread>
#include <vector>
#include <algorithm>

#define CHECK(cmd) do {                                \
    hipError_t e = cmd;                               \
    if (e != hipSuccess) {                            \
        std::cerr << "CUDA Error: " << hipGetErrorString(e) \
                  << " (code " << e << ") at " << __FILE__ << ":" << __LINE__ << std::endl; \
        exit(EXIT_FAILURE);                            \
    }                                                  \
} while(0)

struct DeviceContext {
    int devId;
    char* buf_send;
    char* buf_recv;
};

// Decide repeat count based on message size
int chooseRepeat(size_t bytes) {
    if (bytes <= (1u<<20))   return 2000;   // <= 1 MB
    if (bytes <= (8u<<20))   return 800;
    if (bytes <= (64u<<20))  return 200;
    if (bytes <= (256u<<20)) return 80;
    return 40;
}

// Run one direction copy and measure bandwidth
double runSingleDirection(int srcDev, char* dst, char* src,
                          size_t bytes, int repeat) {
    CHECK(hipSetDevice(srcDev));
    hipStream_t stream;
    CHECK(hipStreamCreate(&stream));
    hipEvent_t start, stop;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));

    CHECK(hipEventRecord(start, stream));
    for (int i = 0; i < repeat; i++) {
        CHECK(hipMemcpyPeerAsync(dst, (srcDev^1), src, srcDev, bytes, stream));
    }
    CHECK(hipEventRecord(stop, stream));
    CHECK(hipEventSynchronize(stop));

    float ms = 0.0f;
    CHECK(hipEventElapsedTime(&ms, start, stop));

    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(stop));
    CHECK(hipStreamDestroy(stream));

    double totalGB = double(bytes) * repeat / 1e9;
    double totalSec = ms / 1e3;
    return totalGB / totalSec;
}

// Bidirectional write test
void runBiWrite(const DeviceContext& d0, const DeviceContext& d1,
                const std::vector<size_t>& sizes, bool print) {
    for (auto bytes : sizes) {
        const int repeat = chooseRepeat(bytes);
        double bw01 = 0.0, bw10 = 0.0;
        std::thread t0([&]{ bw01 = runSingleDirection(d0.devId, d1.buf_recv, d0.buf_send, bytes, repeat); });
        std::thread t1([&]{ bw10 = runSingleDirection(d1.devId, d0.buf_recv, d1.buf_send, bytes, repeat); });
        t0.join(); t1.join();
        if (print) {
            std::cout << "Write: dev0->dev1 | Size = " << (bytes / 1024.0 / 1024.0)
                      << " MB | Bandwidth = " << bw01 << " GB/s\n";
            std::cout << "Write: dev1->dev0 | Size = " << (bytes / 1024.0 / 1024.0)
                      << " MB | Bandwidth = " << bw10 << " GB/s\n";
        }
    }
}

// Bidirectional read test
void runBiRead(const DeviceContext& d0, const DeviceContext& d1,
               const std::vector<size_t>& sizes, bool print) {
    for (auto bytes : sizes) {
        const int repeat = chooseRepeat(bytes);
        double bw01 = 0.0, bw10 = 0.0;
        std::thread t0([&]{ bw01 = runSingleDirection(d1.devId, d0.buf_recv, d1.buf_send, bytes, repeat); });
        std::thread t1([&]{ bw10 = runSingleDirection(d0.devId, d1.buf_recv, d0.buf_send, bytes, repeat); });
        t0.join(); t1.join();
        if (print) {
            std::cout << "Read: dev0<-dev1 | Size = " << (bytes / 1024.0 / 1024.0)
                      << " MB | Bandwidth = " << bw01 << " GB/s\n";
            std::cout << "Read: dev1<-dev0 | Size = " << (bytes / 1024.0 / 1024.0)
                      << " MB | Bandwidth = " << bw10 << " GB/s\n";
        }
    }
}

// Run with warmup wrapper
void runWithWarmup(const char* title,
                   void (*passFn)(const DeviceContext&, const DeviceContext&, const std::vector<size_t>&, bool),
                   const DeviceContext& d0, const DeviceContext& d1,
                   const std::vector<size_t>& sizes) {
    // warmup (not print)
    passFn(d0, d1, sizes, false);
    // real run with output
    std::cout << title << "\n";
    passFn(d0, d1, sizes, true);
}

// Run tests based on type
void runTestsByType(const std::string& testType,
                   const DeviceContext& d0, const DeviceContext& d1,
                   const std::vector<size_t>& various_sizes,
                   const std::vector<size_t>& decode_sizes,
                   const std::vector<size_t>& prefill_sizes) {
    if (testType == "write") {
        // Only run write tests
        runWithWarmup("=== Bidirectional Write (various sizes) ===", runBiWrite, d0, d1, various_sizes);
        std::cout << "\n\n";
        runWithWarmup("=== Bidirectional Write (All-Reduce decode sizes) ===", runBiWrite, d0, d1, decode_sizes);
        std::cout << "\n\n";
        runWithWarmup("=== Bidirectional Write (All-Reduce prefill sizes) ===", runBiWrite, d0, d1, prefill_sizes);
    } else if (testType == "read") {
        // Only run read tests
        runWithWarmup("=== Bidirectional Read  (various sizes) ===", runBiRead,  d0, d1, various_sizes);
        std::cout << "\n\n";
        runWithWarmup("=== Bidirectional Read  (All-Reduce decode sizes) ===", runBiRead,  d0, d1, decode_sizes);
        std::cout << "\n\n";
        runWithWarmup("=== Bidirectional Read  (All-Reduce prefill sizes) ===", runBiRead,  d0, d1, prefill_sizes);
    } else {
        // Run all tests (default behavior)
        runWithWarmup("=== Bidirectional Write (various sizes) ===", runBiWrite, d0, d1, various_sizes);
        runWithWarmup("=== Bidirectional Read  (various sizes) ===", runBiRead,  d0, d1, various_sizes);
        std::cout << "\n\n";

        runWithWarmup("=== Bidirectional Write (All-Reduce decode sizes) ===", runBiWrite, d0, d1, decode_sizes);
        runWithWarmup("=== Bidirectional Read  (All-Reduce decode sizes) ===", runBiRead,  d0, d1, decode_sizes);
        std::cout << "\n\n";

        runWithWarmup("=== Bidirectional Write (All-Reduce prefill sizes) ===", runBiWrite, d0, d1, prefill_sizes);
        runWithWarmup("=== Bidirectional Read  (All-Reduce prefill sizes) ===", runBiRead,  d0, d1, prefill_sizes);
    }
}

void initDevice(DeviceContext& ctx, size_t cap) {
    CHECK(hipSetDevice(ctx.devId));
    CHECK(hipMalloc(&ctx.buf_send, cap));
    CHECK(hipMalloc(&ctx.buf_recv, cap));
    CHECK(hipMemset(ctx.buf_send, 0, cap));
    CHECK(hipMemset(ctx.buf_recv, 0, cap));
}

int main(int argc, char* argv[]) {
    // Parse command line arguments
    std::string testType;
    if (argc == 2) {
        testType = argv[1];
        if (testType != "write" && testType != "read") {
            std::cerr << "Invalid argument. Use: " << argv[0] << " [write|read]\n";
            return EXIT_FAILURE;
        }
    } else if (argc > 2) {
        std::cerr << "Too many arguments. Use: " << argv[0] << " [write|read]\n";
        return EXIT_FAILURE;
    }

    // Message sizes
    std::vector<size_t> various_sizes = { size_t(1)<<20, size_t(8)<<20, size_t(64)<<20, size_t(256)<<20 };
    std::vector<size_t> decode_sizes  = { size_t(8)<<10, size_t(64)<<10 };   // 8KB, 64KB
    std::vector<size_t> prefill_sizes = { size_t(8)<<20, size_t(16)<<20 };   // 8MB, 16MB

    size_t max_needed = 0;
    for (auto v: various_sizes) max_needed = std::max(max_needed, v);
    for (auto v: decode_sizes)  max_needed = std::max(max_needed, v);
    for (auto v: prefill_sizes) max_needed = std::max(max_needed, v);

    DeviceContext d0{0}, d1{1};
    initDevice(d0, max_needed);
    initDevice(d1, max_needed);

    // Enable peer access
    CHECK(hipSetDevice(0));
    CHECK(hipDeviceEnablePeerAccess(1, 0));
    CHECK(hipSetDevice(1));
    CHECK(hipDeviceEnablePeerAccess(0, 0));

    // Run tests based on type
    runTestsByType(testType, d0, d1, various_sizes, decode_sizes, prefill_sizes);

    // Cleanup
    CHECK(hipFree(d0.buf_send));
    CHECK(hipFree(d0.buf_recv));
    CHECK(hipFree(d1.buf_send));
    CHECK(hipFree(d1.buf_recv));
    return 0;
}
    