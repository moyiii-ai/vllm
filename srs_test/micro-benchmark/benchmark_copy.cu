#include <iostream>
#include <cstdlib>
#include <cstdio>
#include <hip/hip_runtime.h>

// Kernel function: Used for data transfer between GPUs
__global__ void copyKernel(int* destination, const int* source, int numElements) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numElements) {
        destination[idx] = source[idx];
    }
}

// Check for CUDA errors
#define CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "Error: " << hipGetErrorString(err) << " in file " << __FILE__ << " at line " << __LINE__ << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

// Helper function: Enable Peer Access between two GPUs (if supported)
void enableGPUPeerAccess(int dev1, int dev2) {
    int can_access = 0;
    CHECK(hipSetDevice(dev1));
    CHECK(hipDeviceCanAccessPeer(&can_access, dev1, dev2));
    
    if (!can_access) {
        std::cerr << "Fatal Error: GPU" << dev1 << " cannot access GPU" << dev2 << " (no peer capability)" << std::endl;
        std::cerr << "Check GPU topology with 'nvidia-smi topo -m' to verify cross-GPU connectivity" << std::endl;
        exit(EXIT_FAILURE);
    }

    CHECK(hipDeviceEnablePeerAccess(dev2, 0));
    CHECK(hipSetDevice(dev2));
    CHECK(hipDeviceEnablePeerAccess(dev1, 0));
    
    std::cout << "Successfully enabled bidirectional peer access between GPU" << dev1 << " and GPU" << dev2 << std::endl;
}

void printUsage(const char* programName) {
    std::cout << "Usage: " << programName << " [direction]" << std::endl;
    std::cout << "Direction parameters:" << std::endl;
    std::cout << "  write   - Transfer data from GPU0 to GPU1" << std::endl;
    std::cout << "  read    - Transfer data from GPU1 to GPU0" << std::endl;
    std::cout << "  both    - Perform write then read (default)" << std::endl;
}

// Write test function: GPU0 -> GPU1, create stream/events inside
void runWriteTest(int* d0_data, int* d1_data, int numElements, int gridSize, int blockSize, int iterations) {
    std::cout << "\nTest: On GPU0, GPU0 writes data to GPU1 using copyKernel..." << std::endl;
    
    // Create stream and events (inside function, bound to GPU0)
    hipStream_t stream;
    hipEvent_t start, stop;
    CHECK(hipSetDevice(0));
    CHECK(hipStreamCreate(&stream));
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));

    // Start timing and run iterations
    CHECK(hipEventRecord(start, stream));
    for (int i = 0; i < iterations; i++) {
        copyKernel<<<gridSize, blockSize, 0, stream>>>(d1_data, d0_data, numElements);
        CHECK(hipGetLastError());
    }
    CHECK(hipEventRecord(stop, stream));
    CHECK(hipEventSynchronize(stop));

    // Calculate and print metrics
    float elapsedMs;
    CHECK(hipEventElapsedTime(&elapsedMs, start, stop));
    double elapsedSec = elapsedMs / 1000.0;
    double dataSizeGB = (numElements * sizeof(int)) / (1024.0 * 1024 * 1024);
    double avgTimePerIter = elapsedSec / iterations;
    double bandwidth = dataSizeGB / avgTimePerIter;
    
    std::cout << "Average bandwidth: " << bandwidth << " GB/s" << std::endl;

    // Cleanup function-specific stream/events
    CHECK(hipStreamDestroy(stream));
    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(stop));
}

// Read test function: GPU1 -> GPU0, create stream/events inside
void runReadTest(int* d0_data, int* d1_data, int numElements, int gridSize, int blockSize, int iterations) {
    std::cout << "\nTest: On GPU0, GPU0 reads data from GPU1 using copyKernel..." << std::endl;
    
    // Create stream and events (inside function, bound to GPU1)
    hipStream_t stream;
    hipEvent_t start, stop;
    CHECK(hipSetDevice(0));
    CHECK(hipStreamCreate(&stream));
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));

    // Start timing and run iterations
    CHECK(hipEventRecord(start, stream));
    for (int i = 0; i < iterations; i++) {
        copyKernel<<<gridSize, blockSize, 0, stream>>>(d0_data, d1_data, numElements);
        CHECK(hipGetLastError());
    }
    CHECK(hipEventRecord(stop, stream));
    CHECK(hipEventSynchronize(stop));

    // Calculate and print metrics
    float elapsedMs;
    CHECK(hipEventElapsedTime(&elapsedMs, start, stop));
    double elapsedSec = elapsedMs / 1000.0;
    double dataSizeGB = (numElements * sizeof(int)) / (1024.0 * 1024 * 1024);
    double avgTimePerIter = elapsedSec / iterations;
    double bandwidth = dataSizeGB / avgTimePerIter;
    
    std::cout << "Average bandwidth: " << bandwidth << " GB/s" << std::endl;

    // Cleanup function-specific stream/events
    CHECK(hipStreamDestroy(stream));
    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(stop));
}

int main(int argc, char* argv[]) {
    // Parse command line arguments (keep simple, no redundant checks)
    const char* mode = "both";
    if (argc == 2)
        mode = argv[1];

    // Configuration (centralized parameters)
    const int iterations = 500;
    const size_t dataSize = 256ULL * 1024 * 1024; // 256MB
    const int numElements = dataSize / sizeof(int);
    const int blockSize = 256;
    const int gridSize = (numElements + blockSize - 1) / blockSize;
    
    // Print basic config
    std::cout << "Data size: " << dataSize / (1024*1024) << "MB" << std::endl;
    std::cout << "Number of elements: " << numElements << std::endl;
    std::cout << "Kernel configuration: " << gridSize << " grids, " << blockSize << " blocks" << std::endl;
    std::cout << "Total iterations: " << iterations << std::endl;
    std::cout << "Test mode: " << mode << std::endl;

    // Step 1: Enable peer access first
    enableGPUPeerAccess(0, 1);

    // Step 2: Allocate memory on both GPUs
    int *d0_data, *d1_data;
    CHECK(hipSetDevice(0));
    CHECK(hipMalloc(&d0_data, dataSize));
    CHECK(hipMemset(d0_data, 0, dataSize));
    
    CHECK(hipSetDevice(1));
    CHECK(hipMalloc(&d1_data, dataSize));
    CHECK(hipMemset(d1_data, 0, dataSize));

    // Wait for user to start counters
    printf("Initialization done! Start counter polling and press...\n");
    getchar();

    // Run tests based on mode (call separate functions)
    if (std::string(mode) == "write" || std::string(mode) == "both") {
        runWriteTest(d0_data, d1_data, numElements, gridSize, blockSize, iterations);
    }
    if (std::string(mode) == "read" || std::string(mode) == "both") {
        runReadTest(d0_data, d1_data, numElements, gridSize, blockSize, iterations);
    }

    // Wait for user to stop counters
    printf("All tests done! Stop counter polling and press...\n");
    getchar();

    // Cleanup global memory
    CHECK(hipSetDevice(0));
    CHECK(hipFree(d0_data));
    
    CHECK(hipSetDevice(1));
    CHECK(hipFree(d1_data));

    std::cout << "\nTests completed" << std::endl;
    return 0;
}